//
// Created by CSWH on 2024/11/17.
//
#include "alloc.hpp"

namespace base {
    void DeviceAllocator::memcpy(const void* src_ptr, void* dest_ptr, size_t byte_size,
                                 MemcpyKind memcpy_kind) const {
        CHECK_NE(src_ptr, nullptr);
        CHECK_NE(dest_ptr, nullptr);
        if (!byte_size) {
            return;
        }

        if (memcpy_kind == MemcpyKind::kMemcpyCPU2CPU) {
            std::memcpy(dest_ptr, src_ptr, byte_size);
        } else if (memcpy_kind == MemcpyKind::kMemcpyCPU2CUDA) {
            hipMemcpy(dest_ptr, src_ptr, byte_size, hipMemcpyHostToDevice);
        } else if (memcpy_kind == MemcpyKind::kMemcpyCUDA2CPU) {
            hipMemcpy(dest_ptr, src_ptr, byte_size, hipMemcpyDeviceToHost);
        } else if (memcpy_kind == MemcpyKind::kMemcpyCUDA2CUDA) {
            hipMemcpy(dest_ptr, src_ptr, byte_size, hipMemcpyDeviceToDevice);
        } else {
            LOG(FATAL) << "Unknown memcpy kind: " << int(memcpy_kind);
        }
    }

    void DeviceAllocator::memset_zero(void* ptr, size_t byte_size) const {
        CHECK(device_type_ != DeviceType::kDeviceUnknown);
        if (device_type_ == DeviceType::kDeviceCPU) {
            std::memset(ptr, 0, byte_size);
        } else {
            hipMemset(ptr, 0, byte_size);
        }
    }
}