# include "alloc.hpp"
# include <stdio.h>

namespace base {
  CUDADeviceAllocator::CUDADeviceAllocator() : DeviceAllocator(DeviceType::kDeviceCUDA) {}

  void* CUDADeviceAllocator::allocate(size_t byte_size) const {
    void* ptr = nullptr;
    hipError_t err = hipMalloc(&ptr, byte_size);
    if(err != hipSuccess) {
      printf("CUDADeviceAllocator::allocate(): hipMalloc failed\n");
      return nullptr;
    }
    return ptr;
  }

  void CUDADeviceAllocator::release(void* ptr) const {
    hipError_t err = hipFree(ptr);
    if(err != hipSuccess) {
      printf("CUDADeviceAllocator::release(): hipFree failed\n");
    }
  }

  std::shared_ptr<CUDADeviceAllocator> CUDADeviceAllocatorFactory::instance = nullptr;
}